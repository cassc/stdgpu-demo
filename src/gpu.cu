#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <stdgpu/unordered_map.cuh>
#include <hip/hip_runtime.h>
#include "main.cuh"


__global__ void gpu_storage_update(stdgpu::unordered_map<int, int> state_map, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        state_map.emplace(idx, idx * 2);
    }
}

struct int_pair_plus
{
    STDGPU_HOST_DEVICE stdgpu::pair<ull, ull>
    operator()(const stdgpu::pair<ull, ull>& lhs, const stdgpu::pair<ull, ull>& rhs) const
    {
        return { lhs.first + rhs.first, lhs.second + rhs.second };
    }
};

int test_gpu() {
    const int size = 1'000'000;

    // GPU unordered map
    stdgpu::unordered_map<int, int> state_map = stdgpu::unordered_map<int, int>::createDeviceObject(size);

    // GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gpu_storage_update<<<(size + 255) / 256, 256>>>(state_map, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "GPU Execution Time: " << elapsedTime << " ms\n";

      // Verify results using thrust
    // Create device vectors to store keys and values
    thrust::device_vector<int> d_keys(size);
    thrust::device_vector<int> d_values(size);


    auto range_map = state_map.device_range();
    stdgpu::pair<ull, ull> sum =
            thrust::reduce(range_map.begin(), range_map.end(), stdgpu::pair<ull, ull>(0, 0), int_pair_plus());

    assert(999999000000 == sum.second);

    return 0;
}
